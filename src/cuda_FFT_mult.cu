
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <vector>
#include <cmath>
#include <stdio.h>
#include <string>
#include <iostream>
#include <openssl/bn.h>
#include <time.h>



/*The multicplication algorithm with cuFFT is from the following source:
Source: https://programmer.group/implementing-large-integer-multiplication-with-cufft.html
The multWithFFT-function is edited by Max & Johannes*/

const auto BATCH = 1;

__global__ void ComplexPointwiseMulAndScale(hipfftComplex *a, hipfftComplex *b, int size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    float scale = 1.0f / (float)size;
    hipfftComplex c;
    for (int i = threadID; i < size; i += numThreads)
    {
        c = hipCmulf(a[i], b[i]);
        b[i] = make_hipFloatComplex(scale*hipCrealf(c), scale*hipCimagf(c));
    }
}

__global__ void ConvertToInt(hipfftReal *a, int size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    auto b = (int*)a;
    for (int i = threadID; i < size; i += numThreads)
        b[i] = static_cast<int>(round(a[i]));
}

std::vector<int> multiply(const std::vector<float> &a, const std::vector<float> &b)
{

	clock_t t;

	t = clock();

    const auto NX = a.size();
    hipfftHandle plan_a, plan_b, plan_c;
    hipfftComplex *data_a, *data_b;
    std::vector<int> c(a.size() + 1);
    c[0] = 0;

    //Allocate graphics card memory and initialize, assuming sizeof(int)==sizeof(float), sizeof(cufftComplex)==2*sizeof(float)
    hipMalloc((void**)&data_a, sizeof(hipfftComplex) * (NX / 2 + 1) * BATCH);
    hipMalloc((void**)&data_b, sizeof(hipfftComplex) * (NX / 2 + 1) * BATCH);
    hipMemcpy(data_a, a.data(), sizeof(float) * a.size(), hipMemcpyHostToDevice);
    hipMemcpy(data_b, b.data(), sizeof(float) * b.size(), hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess) { fprintf(stderr, "Cuda error: Failed to allocate\n"); return c; }

    if (hipfftPlan1d(&plan_a, NX, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS) { fprintf(stderr, "CUFFT error: Plan creation failed"); return c; }
    if (hipfftPlan1d(&plan_b, NX, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS) { fprintf(stderr, "CUFFT error: Plan creation failed"); return c; }
    if (hipfftPlan1d(&plan_c, NX, HIPFFT_C2R, BATCH) != HIPFFT_SUCCESS) { fprintf(stderr, "CUFFT error: Plan creation failed"); return c; }

	t = clock() - t;
	double time_taken_GPU = ((double)t)/CLOCKS_PER_SEC; // in seconds
	printf("Memory and plan: %f s\n", time_taken_GPU);


	t = clock();

    //Converting A(x) to Frequency Domain
    if (hipfftExecR2C(plan_a, (hipfftReal*)data_a, data_a) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return c;
    }

    //Converting B(x) to Frequency Domain
    if (hipfftExecR2C(plan_b, (hipfftReal*)data_b, data_b) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return c;
    }

    //Point multiplication
    ComplexPointwiseMulAndScale<<<NX / 256 + 1, 256>>>(data_a, data_b, NX);

    //Converting C(x) back to time domain
    if (hipfftExecC2R(plan_c, data_b, (hipfftReal*)data_b) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: ExecC2R Forward failed");
        return c;
    }

    //Converting the results of floating-point numbers to integers
    ConvertToInt<<<NX / 256 + 1, 256>>>((hipfftReal*)data_b, NX);

    if (hipDeviceSynchronize() != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Failed to synchronize\n");
        return c;
    }

    t = clock() - t;
    time_taken_GPU = ((double)t)/CLOCKS_PER_SEC; // in seconds
    printf("Calc: %f s\n", time_taken_GPU);


    t = clock();
    hipMemcpy(&c[1], data_b, sizeof(float) * b.size(), hipMemcpyDeviceToHost);

    hipfftDestroy(plan_a);
    hipfftDestroy(plan_b);
    hipfftDestroy(plan_c);
    hipFree(data_a);
    hipFree(data_b);

    t = clock() - t;
    time_taken_GPU = ((double)t)/CLOCKS_PER_SEC; // in seconds
    printf("Cleaning: %f s\n", time_taken_GPU);

    return c;
}

void print(std::vector<float> const &input)
{
	for (int i = 0; i < input.size(); i++) {
		std::cout << input.at(i) << ' ';
	}
}

extern "C" void multWithFFT(BIGNUM* a, BIGNUM *b, BIGNUM **c)
{

    const int base = 10;
    char* a_String = BN_bn2dec(a);
    char* b_String = BN_bn2dec(b);

    int lengthA = strlen(a_String);
    int lengthB = strlen(b_String);

	printf("a: %s\n",a_String);
	printf("b: %s\n",b_String);

    //length of multiplication result has the size of the sum of the two factors
    int result_length = lengthA + lengthB;

    //factors are stored in these vectors
    std::vector<float> av{};
    std::vector<float> bv{};

    //fill vectors step by step
    for(int i=0; i<lengthA; ++i){
    	av.push_back((float)(a_String[i])-'0');
    }
    for(int i=0; i<lengthB; ++i){
    	bv.push_back((float)(b_String[i])-'0');
    }

    //vectors need to be same size
    while (av.size() != result_length){
    	 av.insert(av.begin(),(float) 0);
    }
    while (bv.size() != result_length){
    	bv.insert(bv.begin(),(float) 0);
    }

	clock_t t;

	t = clock();
    //call cuda-kernel-function
    std::vector<int> cv = multiply(av, bv);
	t = clock() - t;
	double time_taken_GPU = ((double)t)/CLOCKS_PER_SEC; // in seconds
	printf("Cuda_multiply: %f s\n", time_taken_GPU);

    //Processing carry
    for (int i = cv.size() - 1; i > 0; i--)
    {
        if (cv[i] >= base)
        {
            cv[i - 1] += cv[i] / base;
            cv[i] %= base;
        }
    }


    //Remove excess zeros
    cv.pop_back();
    auto i = 0;

    //For some multiplications the result has a zero as a first digit (for example 999*1 = 999 will be 0999)
    if (cv[0] == 0)
        i++;

    //If i++ will be executed the array still has to begin at element tmp[0]
    int k = 0;

	char tmp[cv.size()];

	//convert integer vector to string
    for (; i < cv.size(); i++){
    	tmp[k] = (char) cv.at(i) + '0';
    	k++;
    }
    tmp[k] = '\0';

    //transfer result to cuda_mult.c
    memcpy(*c,tmp,sizeof(tmp));

    return;
}
